#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

typedef unsigned int    uint32;
typedef int             int32;
typedef unsigned char   uint8;
typedef struct __uint32
{
	uint8 r;
	uint8 g;
	uint8 b;
	uint8 a;
};


extern "C"
__global__ void cutPicture(uint32 * srcImage, int srcWidth, int srcHeight, uint32 * dstImage, int cx, int cy, int dstWidth, int dstHeight)
{
	int32 x, y;
	int srcPitch = srcWidth * 4;
	int dstPitch = dstWidth * 4;
	__uint32 *srcImageU8 = (__uint32 *)srcImage;
	__uint32 *dstImageU8 = (__uint32 *)dstImage;

	x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	y = blockIdx.y *  blockDim.y + threadIdx.y;

	if ((x-1) >= dstWidth)
		return; //x = width - 1;

	if ((y-1) >= dstHeight)
		return; // y = height - 1;

	dstImageU8[y * dstWidth + x].b = srcImageU8[(y + cy) * srcWidth + x + cx].b;
	dstImageU8[y * dstWidth + x].g = srcImageU8[(y + cy) * srcWidth + x + cx].g;
	dstImageU8[y * dstWidth + x].r = srcImageU8[(y + cy) * srcWidth + x + cx].r;
	dstImageU8[y * dstWidth + x].a = srcImageU8[(y + cy) * srcWidth + x + cx].a;
	dstImageU8[y * dstWidth + x + 1].b = srcImageU8[(y + cy) * srcWidth + x + cx + 1].b;
	dstImageU8[y * dstWidth + x + 1].g = srcImageU8[(y + cy) * srcWidth + x + cx + 1].g;
	dstImageU8[y * dstWidth + x + 1].r = srcImageU8[(y + cy) * srcWidth + x + cx + 1].r;
	dstImageU8[y * dstWidth + x + 1].a = srcImageU8[(y + cy) * srcWidth + x + cx + 1].a;
}

int MattingPicture(void * srcImage, int srcWidth, int srcHeight, void * dstImage, int x, int y, int dstWidth, int dstHeight)
{
	hipError_t cudaStatus;
	dim3 block(32, 16, 1);
	dim3 grid((dstWidth + (2 * block.x - 1)) / (2 * block.x), (dstHeight + (block.y - 1)) / block.y, 1);
	cutPicture << <grid, block, 0 >> > ((uint32*)srcImage, srcWidth, srcHeight, (uint32*)dstImage, x, y, dstWidth, dstHeight);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	return (int)cudaStatus;
}